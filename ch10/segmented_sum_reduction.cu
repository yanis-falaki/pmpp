
#include <hip/hip_runtime.h>
#define BLOCK_DIM 1024
__global__ void SegmentedSumReductionKernel(float* input, float* output) {
    __shared__ float input_s[BLOCK_DIM];

    unsigned int segment = 2*blockDim.x*blockIdx.x;
    unsigned int i = segment + threadIdx.x;
    unsigned int t = threadIdx.x;

    input_s[t] = input[i] + input[i + BLOCK_DIM];
    for (unsigned int stride = blockDim.x/2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (t < stride) {
            input_s[t] += input_s[t + stride];
        }
    }

    if (t == 0) {
        atomicAdd(output, input_s[0]);
    }
}


#define COARSE_FACTOR 3
__global__ void CoarsenedSumReductionKernel(float* input, float* output) {
    __shared__ float input_s[BLOCK_DIM];

    unsigned int segment = COARSE_FACTOR*2*blockDim.x*blockIdx.x;
    unsigned int i = segment + threadIdx.x;
    unsigned int t = threadIdx.x;

    float sum = input[i];
    for (unsigned int tile = 1; tile < COARSE_FACTOR*2; ++tile) {
        sum += input[i + tile*BLOCK_DIM];
    }
    input_s[t] = sum;

    for (unsigned int stride = blockDim.x/2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (t < stride) {
            input_s[t] += input_s[t + stride];
        }
    }

    if (t == 0) {
        atomicAdd(output, input_s[0]);
    }
}