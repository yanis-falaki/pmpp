
#include <hip/hip_runtime.h>
__constant__ int c0 = 0;
__constant__ int c1 = 1;
__constant__ int c2 = 1;
__constant__ int c3 = 1;
__constant__ int c4 = 1;
__constant__ int c5 = 1;
__constant__ int c6 = 1;


__global__ void stencil_kernel(float* in, float* out, unsigned int N) {

    int i = blockIdx.z*blockDim.z + threadIdx.z;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.x*blockDim.x + threadIdx.x;

    if (i >= 1 && i < N - 1 && j >= 1 && j < N-1 && k >= 1 && k < N-1) {
        out[i*N*N + j*N + k] = c0*in[i*N*N + j*N + k]
                                + c1*in[i*N*N + j*N + (k-1)]
                                + c2*in[i*N*N + j*N + (k+1)]
                                + c3*in[i*N*N + (j-1)*N + k]
                                + c4*in[i*N*N + (j+1)*N + k]
                                + c5*in[(i-1)*N*N + j*N + k]
                                + c6*in[(i+1)*N*N + j*N + k];
    }
}