
#include <hip/hip_runtime.h>
#include <iostream>
#define NUM_BINS 7 // ceil(26/4) = 7

__global__ void histo_private_coarsened_interleaved_kernel(char* data, unsigned int length, unsigned int* histo) {
    // Initialize privatized bins
    __shared__ unsigned int histo_s[NUM_BINS];
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        histo_s[bin] = 0u;
    }
    __syncthreads();

    // Histogram
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    for (unsigned int i = tid; i < length; i += blockDim.x*gridDim.x) {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) {
            atomicAdd(&(histo_s[alphabet_position/4]), 1);
        }
    }
    __syncthreads();
    
    // Commit to global memory
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        unsigned int binValue = histo_s[bin];
        if (binValue > 0) {
            atomicAdd(&(histo[bin]), binValue);
        }
    }
}